#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <string>
#include "hip/hip_runtime.h"

#include "classes/Particle.hpp"
#include "classes/Vector.hpp"
#include "classes/Grid.hpp"
#include "kernels/test.cu"

#define HANDLE_CUDA_ERROR(ERROR) if ((ERROR) != hipSuccess) { \
		fprintf(stderr, "Cuda memory error"); \
		return 1; \
		} \


__host__  void writeToFile(Grid<Particle> * grid, std::ofstream &file) {

	int size = grid->getRows() * grid->getCols();

	file << size << std::endl;
	file << std::endl;

	for (int row = 0; row < grid->getRows(); row++)
		for (int col = 0; col < grid->getCols(); col++) {
			Vec3 vec = grid->get(row, col).position;
			file << vec.x << " " << vec.y << " " << vec.z << std::endl;
		}
}

//A test to show how to work with classes and CUDA
//See Grid.hpp and test.cu
int deviceCompatibleClassExample() {
	Grid<int> * g = new Grid<int>(1, 1);
	Grid<int> * g_device = Grid<int>::gridcpy(g, Grid<int>::UPLOAD);

	change << <1, 1 >> > (g_device, 0, 0, 4);

	Grid<int> * d = Grid<int>::gridcpy(g_device, Grid<int>::DOWNLOAD);

	std::cout << d->get(0, 0) << std::endl;

	return 0;
}

int vectorsExample() {
	Vec3 vec1cpu[100], vec2cpu[100], vec3cpu[100];

	// Init arrays with whatever
	for (int i = 0; i < 100; i++) vec1cpu[i] = { 2.0f * i, 7.0f * i, 13.0f * i };
	for (int i = 0; i < 100; i++) vec2cpu[i] = { 3.0f * i, 11.0f * i, 17.0f * i };

	Vec3 *vec1 = { 0 }, *vec2 = { 0 }, *vec3 = { 0 };
	size_t vecArrSize = 100 * sizeof(Vec3);

	// Alloc 3, 100 Vec3 arrays
	HANDLE_CUDA_ERROR(hipMalloc((void **)&vec1, vecArrSize));
	HANDLE_CUDA_ERROR(hipMalloc((void **)&vec2, vecArrSize));
	HANDLE_CUDA_ERROR(hipMalloc((void **)&vec3, vecArrSize));

	// Memcpy vec1, vec2
	HANDLE_CUDA_ERROR(hipMemcpy(vec1, vec1cpu, vecArrSize, hipMemcpyHostToDevice));
	HANDLE_CUDA_ERROR(hipMemcpy(vec2, vec2cpu, vecArrSize, hipMemcpyHostToDevice));

	// Add vectors in parallel.
	sum << <1, 100 >> > (vec1, vec2, vec3);

	// Sync device
	HANDLE_CUDA_ERROR(hipGetLastError());
	HANDLE_CUDA_ERROR(hipDeviceSynchronize());
	HANDLE_CUDA_ERROR(hipMemcpy(vec3cpu, vec3, vecArrSize, hipMemcpyDeviceToHost));
	// Copy result to CPU

	for (int i = 0; i < 100; i++) {
		printf("Vec3 {%f, %f, %f} \n", vec3cpu[i].x, vec3cpu[i].y, vec3cpu[i].z);
	}

	hipFree(vec1);
	hipFree(vec2);
	hipFree(vec3);

	return 0;
}

#define DUMP_FILE "particles.dump"

int main(){
	hipError_t status;

	int rows = 100, cols = 100;
	int ticks = 100;

	Grid<Particle> * g = new Grid<Particle>(rows, cols);
	Grid<Particle> * g_device = Grid<Particle>::gridcpy(g, Grid<Particle>::UPLOAD);

	dim3 dimBlock = dim3(10, 10);
	int yBlocks = cols / dimBlock.y + ((cols%dimBlock.y) == 0 ? 0 : 1);
	int xBlocks = rows / dimBlock.x + ((rows%dimBlock.x) == 0 ? 0 : 1);
	dim3 dimGrid = dim3(xBlocks, yBlocks);


	initializePositions <<<dimGrid, dimBlock >>> (g_device, 1);
	Grid<Particle> * d;
	d = Grid<Particle>::gridcpy(g_device, Grid<Particle>::DOWNLOAD);
	
	std::ofstream file(DUMP_FILE);
	for (int i = 0; i < ticks; i++) {
		//Try moveDownwardsCool!
		moveDownwards << <dimGrid, dimBlock >> > (g_device, 1);
		d = Grid<Particle>::gridcpy(g_device, Grid<Particle>::DOWNLOAD);

		writeToFile(d, file);
	}
	file.close();
	
	status = hipDeviceReset();
	return 0;
}