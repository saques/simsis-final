#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <string>
#include <chrono>
#include <thread>
#include <queue>
#include <mutex>
#include <Windows.h>
#include "hip/hip_runtime.h"

#include "classes/Particle.hpp"
#include "classes/Vector.hpp"
#include "classes/Grid.hpp"
#include "kernels/test.cu"
#include "kernels/grid_kernels.cu"

#define HANDLE_CUDA_ERROR(ERROR) if ((ERROR) != hipSuccess) { \
		fprintf(stderr, "Cuda memory error"); \
		return 1; \
		} \

#define FRAME_RATE 60

__host__  void writeToFile(Grid<Particle> * grid, std::ofstream &file, Particle * big, int size) {
	int s = grid->getRows() * grid->getCols() + size;
	char * buf = new char[s * 50 * 4];
	char * bufp = (char *) buf;

	int len = sprintf(bufp, "%d \n\n", s);
	bufp += len;

	for (int i = 0; i < size; i++) {
		Particle p = big[i];
		Vec3 vec = p.position;
		int len = sprintf(bufp, "%f %f %f %f\n", vec.x, vec.y, vec.z, p.radius);
		bufp += len;
	}

	for (int row = 0; row < grid->getRows(); row++)
		for (int col = 0; col < grid->getCols(); col++) {
			Particle p = grid->get(row, col);
			Vec3 vec = p.position;
			int len = sprintf(bufp, "%f %f %f %f\n", vec.x, vec.y, vec.z, p.radius);
			bufp += len;
		}
	file.write(buf, bufp - buf);
	delete[] buf;
}

__host__ void computeBigMassForces(Particle * p_host, int p_size, Grid<Particle> * g_device, float g, float separation, float kn, float kt, float natural, float kbig) {

	Particle * p_device;
	hipMalloc((void **)&(p_device), sizeof(Particle)*p_size);
	hipMemcpy(p_device, p_host, sizeof(Particle)*p_size, hipMemcpyHostToDevice);

	//Reset forces in particles resetBigParticles(Particle * particles, int size, float g)
	resetBigParticles<<<1, p_size>>>(p_device, p_size, g);

	interactBigParticles << <1, p_size >> > (p_device, p_size, natural, kbig);

	//Interact grid and particle
	int x_start = (int)((p_host->position.x - p_host->radius) / separation);
	int y_start = (int)((p_host->position.y - p_host->radius) / separation);

	int x_end = x_start + (int)(2 * p_host->radius / separation);
	int y_end = y_start + (int)(2 * p_host->radius / separation);


	dim3 dimBlock = dim3(10, 10);
	int yBlocks = (y_end - y_start) / dimBlock.y + (((y_end - y_start) % dimBlock.y) == 0 ? 0 : 1);
	int xBlocks = (x_end - x_start) / dimBlock.x + (((x_end - x_start) % dimBlock.x) == 0 ? 0 : 1);
	dim3 dimGrid = dim3(xBlocks, yBlocks);

	//__global__ void interactGridAndParticle(Grid<Particle> * grid, Particle * big, int start_x, int start_y, int end_x, int end_y, float kn) {

	for(int i = 0; i < p_size; i++)
		interactGridAndParticle << <dimGrid, dimBlock >> > (g_device, p_device, i, x_start, y_start, x_end, y_end, kn, kt);

	hipMemcpy(p_host, p_device, sizeof(Particle)*p_size, hipMemcpyDeviceToHost);
	hipFree(p_device);
}


__host__ void updateEulerBigMass(Particle * p_host, int p_size, float delta_t) {

	Particle * p_device;
	hipMalloc((void **)&(p_device), sizeof(Particle)*p_size);
	hipMemcpy(p_device, p_host, sizeof(Particle)*p_size, hipMemcpyHostToDevice);

	updateEulerBigParticles << <1, p_size >> > (p_device, p_size, delta_t);

	hipMemcpy(p_host, p_device, sizeof(Particle)*p_size, hipMemcpyDeviceToHost);
	hipFree(p_device);
}





#define DUMP_FOLDER "dump"
#define THREAD_COUNT 8
int main(){
	
	hipError_t status;

	float simulation_t = 5;
	float delta_t = 0.0001f;
	int rows = 75, cols = 75;
	int frame_rate = 60;
	float separation = 0.05, mass = 0.005, radius = 0.01, g_earth = 9.81, k = 3000;
	int skip_x = 1, skip_y = 1;

	float big_mass = 3, big_radius = 0.15, kn = 1E5, kt = 1E3, separation_big = big_radius, kbig = 1E8;
	Vec3 big_init = { rows/2*separation, rows/2*separation, 3 };

	int ticks = simulation_t/delta_t;
	int dump_each = (int) ((1.0 / frame_rate) / delta_t);

	Grid<Particle> * g = new Grid<Particle>(rows, cols);
	Grid<Particle> * g_device = Grid<Particle>::gridcpy(g, Grid<Particle>::UPLOAD);

	int big_size = 2;
	Particle * big = newParticles(big_init, big_mass, big_radius, big_size, separation_big);

	dim3 dimBlock = dim3(10, 10);
	int yBlocks = cols / dimBlock.y + ((cols%dimBlock.y) == 0 ? 0 : 1);
	int xBlocks = rows / dimBlock.x + ((rows%dimBlock.x) == 0 ? 0 : 1);
	dim3 dimGrid = dim3(xBlocks, yBlocks);

	std::chrono::high_resolution_clock clock;
	
	initializePositions <<<dimGrid, dimBlock >>> (g_device, separation, mass, radius);
	Grid<Particle> * d;
	
	auto pre = clock.now();

	std::queue<Grid<Particle>*> q;
	int count = 0;
	std::mutex m;

	CreateDirectory(DUMP_FOLDER, nullptr);
	auto writer = [&q, ticks, dump_each, &count, &m, big, big_size] {
		int total_count = ticks / dump_each;
		while (count < total_count) {
			Grid<Particle>* el = nullptr;
			int c = 0;
			m.lock();
			if (!q.empty()) {
				el = q.front();
				q.pop();
				c = count;
				count++;
			}
			m.unlock();
			if (el != nullptr) {
				std::ofstream file(std::string(DUMP_FOLDER) + "/" + std::to_string(c) + ".dump");
				writeToFile(el, file, big, big_size);
				delete el;
				file.close();
			}
			else {
				std::this_thread::sleep_for(std::chrono::milliseconds(100));
			}
		}
	};

	std::thread t[THREAD_COUNT];
	for (int i = 0; i < THREAD_COUNT; i++) t[i] = std::thread(writer);

	for (int i = 0; i < ticks; i++) {
		//Try moveDownwardsCool!
		/*reset << <dimGrid, dimBlock >> > (g_device, g_earth, skip_x, skip_y);
		gridElasticForce << <dimGrid, dimBlock >> > (g_device, k, separation, skip_x, skip_y);
		updateEuler << <dimGrid, dimBlock >> > (g_device, delta_t, skip_x, skip_y);*/
		
		verletPositions << <dimGrid, dimBlock >> > (g_device, delta_t, skip_x, skip_y);
		reset << <dimGrid, dimBlock >> > (g_device, g_earth, skip_x, skip_y);
		gridElasticForce << <dimGrid, dimBlock >> > (g_device, k, separation, skip_x, skip_y);
		computeBigMassForces(big, big_size, g_device, g_earth, separation, kn, kt, separation_big, kbig);
		verletVelocities << <dimGrid, dimBlock >> > (g_device, delta_t, skip_x, skip_y);
		updateEulerBigMass(big, big_size, delta_t);

		if (i % dump_each == 0) {
			// Dump to file
			d = Grid<Particle>::gridcpy(g_device, Grid<Particle>::DOWNLOAD);
			m.lock();
			q.push(d);
			m.unlock();
		}
	}
	printf("Waiting for disk operations...");
	auto waiting = clock.now();


	for (int i = 0; i < THREAD_COUNT; i++) t[i].join();

	auto waited = std::chrono::duration_cast<std::chrono::milliseconds>(clock.now() - waiting);
	printf("Wasted %f seconds writing\n", waited / 1000.0);
	auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(clock.now() - pre);
	printf("Millis elapsed: %d\n", elapsed.count());

	
	status = hipDeviceReset();
	return 0;
}